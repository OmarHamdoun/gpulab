#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
 *
 * time:    winter term 2012/13 / March 11-18, 2013
 *
 * project: superresolution
 * file:    linearoperations.cu
 *
 *
 * implement all functions with ### implement me ### in the function body
 \****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */

#include <auxiliary/cuda_basic.cuh>

hipChannelFormatDesc linearoperation_float_tex =
		hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;

#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];

void setTexturesLinearOperations(int mode)
{
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if (mode == 0)
		tex_linearoperation.filterMode = hipFilterModePoint;
	else
		tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}

#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif

#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*) address;
	unsigned int old = *address_as_ull, assumed;
	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	} while (assumed != old);
	return __int_as_float(old);
}

#endif

void backwardRegistrationBilinearValueTex(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g, float value,
		int nx, int ny, int pitchf1_in, int pitchf1_out, float hx, float hy)
{
	// ### Implement me ###
}

// gpu warping kernel
__global__ void backwardRegistrationBilinearFunctionGlobalGpu(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// check if x is within the boundaries
	if (x < nx && y < ny)
	{
		const float xx = (float) x + flow1_g[y * pitchf1_in + x] / hx;
		const float yy = (float) y + flow2_g[y * pitchf1_in + x] / hy;

		int xxFloor = (int) floor(xx);
		int yyFloor = (int) floor(yy);

		int xxCeil = xxFloor == nx - 1 ? xxFloor : xxFloor + 1;
		int yyCeil = yyFloor == ny - 1 ? yyFloor : yyFloor + 1;

		float xxRest = xx - (float) xxFloor;
		float yyRest = yy - (float) yyFloor;

		//same weird expression as in cpp
		out_g[y * pitchf1_out + x] =
		(xx < 0.0f || yy < 0.0f || xx > (float) (nx - 1)
						|| yy > (float) (ny - 1)) ? constant_g[y * pitchf1_in + x] :
						(1.0f - xxRest) * (1.0f - yyRest)* in_g[yyFloor * pitchf1_in + xxFloor]
								+ xxRest * (1.0f - yyRest)* in_g[yyFloor * pitchf1_in + xxCeil]
								+ (1.0f - xxRest) * yyRest* in_g[yyCeil * pitchf1_in + xxFloor]
								+ xxRest * yyRest* in_g[yyCeil * pitchf1_in + xxCeil];

	}
}

// initialize cuda warping kernel
void backwardRegistrationBilinearFunctionGlobal(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	//same construction as in main flow to compute block and grid size
	int ngx = (nx % LO_BW) ? ((nx / LO_BW) + 1) : (nx / LO_BW);
	int ngy = (ny % LO_BH) ? ((ny / LO_BH) + 1) : (ny / LO_BH);

	dim3 dimGrid(ngx, ngy);
	dim3 dimBlock(LO_BW, LO_BH);

	//call warp method on gpu
	backwardRegistrationBilinearFunctionGlobalGpu<<<dimGrid, dimBlock>>>(in_g,
			flow1_g, flow2_g, out_g, constant_g, nx, ny, pitchf1_in,
			pitchf1_out, hx, hy);
}

void backwardRegistrationBilinearFunctionTex(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	// ### Implement me, if you want ###
}

void forewardRegistrationBilinearAtomic(const float *flow1_g,
		const float *flow2_g, const float *in_g, float *out_g, int nx, int ny,
		int pitchf1)
{
	// ### Implement me ###
}

void gaussBlurSeparateMirrorGpu(float *in_g, float *out_g, int nx, int ny,
		int pitchf1, float sigmax, float sigmay, int radius, float *temp_g,
		float *mask)
{
	// ### Implement me ###
}

__global__ void resampleAreaParallelSeparate_x
(
		const float * in_g,
		float * out_g,
		int nx,
		int ny,
		float hx,
		int pitchf1_in,
		float factor = 0.0f
)
{
	if( factor == 0.0f ) { factor = 1/hx; }

	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;

	int index = ix + iy * pitchf1_in; // global index for out image

	if( ix < nx && iy < ny)
	{
		// initialising out
		out_g[ index ] = 0.0f;

		float px = (float)ix * hx;

		float left = ceil(px) - px;
		if(left > hx) left = hx;

		float midx = hx - left;
		float right = midx - floorf(midx);
		midx = midx - right;

		if( left > 0.0f )
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int) floor(px) ] * left * factor; // look out for conversion of coordinates
			px += 1.0f;
		}
		while(midx > 0.0f)
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * factor;
			px += 1.0f;
			midx -= 1.0f;
		}
		if(right > RESAMPLE_EPSILON)
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * right * factor;
		}
	}
}

__global__ void resampleAreaParallelSeparate_y
(
		const float * in_g,
		float * out_g,
		int nx,
		int ny,
		float hy,
		int pitchf1_out,
		float factor = 0.0f // need
)
{
	if(factor == 0.0f) factor = 1.0f/hy;

	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;

	int index = ix + iy * pitchf1_out; // global index for out image
									  // used pitch instead  of blockDim.x

	if( ix < nx && iy < ny ) // guards
	{
		out_g[index] = 0.0f;

		float py = (float)iy * hy;
		float top = ceil(py) - py;

		if(top > hy) top = hy;
		float midy = hy - top;

		float bottom = midy - floorf(midy);
		midy = midy - bottom;

		if(top > 0.0f)
		{
			// using pitch for helper array since these all arrays have same pitch
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * top * factor;
			py += 1.0f;
		}
		while(midy > 0.0f)
		{
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * factor;
			py += 1.0f;
			midy -= 1.0f;
		}
		if(bottom > RESAMPLE_EPSILON)
		{
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * bottom * factor;
		}
	}
}

void resampleAreaParallelSeparate(const float *in_g, float *out_g, int nx_in,
		int ny_in, int pitchf1_in, int nx_out, int ny_out, int pitchf1_out,
		float *help_g, float scalefactor)
{
	// helper array is already allocated on the GPU as _b1, now help_g

	// can reduce no of blocks for first pass
	dim3 dimGrid((int)ceil((float)nx_out/LO_BW), (int)ceil((float)ny_out/LO_BH));
	dim3 dimBlock(LO_BW,LO_BH);

	float hx = (float) nx_in/ (float) nx_out;
	float factor = (float)(nx_out)/(float)(nx_in);
	resampleAreaParallelSeparate_x<<< dimGrid, dimBlock >>>( in_g, help_g, nx_out, ny_in, hx, pitchf1_in, factor);

	float hy = (float)(ny_in)/(float)(ny_out);
	factor = scalefactor*(float)(ny_out)/(float)(ny_in);
	resampleAreaParallelSeparate_y<<< dimGrid, dimBlock >>>( help_g, out_g, nx_out, ny_out, hy, pitchf1_out, factor );
}

void resampleAreaParallelSeparateAdjoined(const float *in_g, float *out_g,
		int nx_in, int ny_in, int pitchf1_in, int nx_out, int ny_out,
		int pitchf1_out, float *help_g, float scalefactor)
{
	// ### Implement me ###
}

__global__ void addKernel(const float *increment_g, float *accumulator_g,
		int nx, int ny, int pitchf1)
{
	// ### Implement me ###
}

__global__ void subKernel(const float *increment_g, float *accumulator_g,
		int nx, int ny, int pitchf1)
{
	// ### Implement me ###
}

__global__ void setKernel(float *field_g, int nx, int ny, int pitchf1,
		float value)
{
	// ### Implement me ###
}

