#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    linearoperations.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */


#include <auxiliary/cuda_basic.cuh>
#include <auxiliary/debug.hpp>

hipChannelFormatDesc linearoperation_float_tex = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;


#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];


void setTexturesLinearOperations(int mode){
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if(mode == 0)tex_linearoperation.filterMode = hipFilterModePoint;
	else tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}


#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif


#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*)address;
	unsigned int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	}	while (assumed != old);
	return __int_as_float(old);
}

#endif




void backwardRegistrationBilinearValueTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		float value,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me ###
}

void backwardRegistrationBilinearFunctionGlobal
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me ###
}

void backwardRegistrationBilinearFunctionTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me, if you want ###
}




void forewardRegistrationBilinearAtomic
(
		const float *flow1_g,
		const float *flow2_g,
		const float *in_g,
		float       *out_g,
		int         nx,
		int         ny,
		int         pitchf1
)
{
	// ### Implement me ###
}




void gaussBlurSeparateMirrorGpu
(
		float *in_g,
		float *out_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float sigmax,
		float sigmay,
		int   radius,
		float *temp_g,
		float *mask
)
{
	// ### Implement me ###
}

__global__ void resampleAreaParallelSeparate_x
(
		const float * in_g,
		float * out_g,
		int nx,
		int ny,
		float hx,
		int pitchf1_in,
		float factor = 0.0f
)
{
	if( factor == 0.0f ) { factor = 1/hx; }
	
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	
	int index = ix + iy * pitchf1_in; // global index for out image
							
	if( ix < nx && iy < ny)
	{
		// initialising out
		out_g[ index ] = 0.0f;
	
		float px = (float)ix * hx;
	
		float left = ceil(px) - px;	
		if(left > hx) left = hx;
	
		float midx = hx - left;
		float right = midx - floorf(midx);
		midx = midx - right;
	
		if( left > 0.0f )
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int) floor(px) ] * left * factor; // look out for conversion of coordinates
			px += 1.0f;
		}
		while(midx > 0.0f)
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * factor;
			px += 1.0f;
			midx -= 1.0f;
		}
		if(right > RESAMPLE_EPSILON)	
		{
			// using pitchf1_in instead of nx_orig in original code
			out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * right * factor;
		}
	}
}

__global__ void resampleAreaParallelSeparate_y
(
		const float * in_g,
		float * out_g,
		int nx,
		int ny,
		float hy,	
		int pitchf1_out,
		float factor = 0.0f // need
)
{
	if(factor == 0.0f) factor = 1.0f/hy;

	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	
	int index = ix + iy * pitchf1_out; // global index for out image
									  // used pitch instead  of blockDim.x
		
	if( ix < nx && iy < ny ) // guards
	{
		out_g[index] = 0.0f;
		
		float py = (float)iy * hy;
		float top = ceil(py) - py;
		
		if(top > hy) top = hy;
		float midy = hy - top;
		
		float bottom = midy - floorf(midy);
		midy = midy - bottom;
		
		if(top > 0.0f)
		{
			// using pitch for helper array since these all arrays have same pitch
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * top * factor;
			py += 1.0f;
		}
		while(midy > 0.0f)
		{
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * factor;
			py += 1.0f;
			midy -= 1.0f;
		}
		if(bottom > RESAMPLE_EPSILON)
		{
			out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * bottom * factor;
		}					
	}	
}

void resampleAreaParallelSeparate
(
		float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
	
	// helper array is already allocated on the GPU as _b1, now help_g
	
	// can reduce no of blocks for first pass
	dim3 dimGrid((int)ceil((float)nx_out/LO_BW), (int)ceil((float)ny_out/LO_BH)); 
	dim3 dimBlock(LO_BW,LO_BH);
		
	float hx = (float) nx_in/ (float) nx_out; 
	float factor = (float)(nx_out)/(float)(nx_in);		
	resampleAreaParallelSeparate_x<<< dimGrid, dimBlock >>>( in_g, help_g, nx_out, ny_in, hx, pitchf1_in, factor);
		
	float hy = (float)(ny_in)/(float)(ny_out);
	factor = scalefactor*(float)(ny_out)/(float)(ny_in);
	resampleAreaParallelSeparate_y<<< dimGrid, dimBlock >>>( help_g, out_g, nx_out, ny_out, hy, pitchf1_out, factor );		
}

void resampleAreaParallelSeparateAdjoined
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}


__global__ void addKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	// ### Implement me ###
}

__global__ void subKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	// ### Implement me ###
}

__global__ void setKernel
(
		float *field_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float value
)
{
	// ### Implement me ###
}

