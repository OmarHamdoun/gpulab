#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
 *
 * time:    winter term 2012/13 / March 11-18, 2013
 *
 * project: superresolution
 * file:    linearoperations.cu
 *
 *
 * implement all functions with ### implement me ### in the function body
 \****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */

#include <auxiliary/cuda_basic.cuh>
#include <iostream>

hipChannelFormatDesc linearoperation_float_tex =
		hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;

#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];

void setTexturesLinearOperations(int mode)
{
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if (mode == 0)
		tex_linearoperation.filterMode = hipFilterModePoint;
	else
		tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}

#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif

#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*) address;
	unsigned int old = *address_as_ull, assumed;
	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	} while (assumed != old);
	return __int_as_float(old);
}

#endif

void backwardRegistrationBilinearValueTex(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g, float value,
		int nx, int ny, int pitchf1_in, int pitchf1_out, float hx, float hy)
{
	// ### Implement me ###
}

// gpu warping kernel
__global__ void backwardRegistrationBilinearFunctionGlobalGpu(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// check if x is within the boundaries
	if (x < nx && y < ny)
	{
		const float xx = (float) x + flow1_g[y * pitchf1_in + x] / hx;
		const float yy = (float) y + flow2_g[y * pitchf1_in + x] / hy;

		int xxFloor = (int) floor(xx);
		int yyFloor = (int) floor(yy);

		int xxCeil = xxFloor == nx - 1 ? xxFloor : xxFloor + 1;
		int yyCeil = yyFloor == ny - 1 ? yyFloor : yyFloor + 1;

		float xxRest = xx - (float) xxFloor;
		float yyRest = yy - (float) yyFloor;

		//same weird expression as in cpp
		out_g[y * pitchf1_out + x] =
				(xx < 0.0f || yy < 0.0f || xx > (float) (nx - 1)
						|| yy > (float) (ny - 1)) ?
						constant_g[y * pitchf1_in + x] :
						(1.0f - xxRest) * (1.0f - yyRest)
								* in_g[yyFloor * pitchf1_in + xxFloor]
								+ xxRest * (1.0f - yyRest)
										* in_g[yyFloor * pitchf1_in + xxCeil]
								+ (1.0f - xxRest) * yyRest
										* in_g[yyCeil * pitchf1_in + xxFloor]
								+ xxRest * yyRest
										* in_g[yyCeil * pitchf1_in + xxCeil];

	}
}

// initialize cuda warping kernel
void backwardRegistrationBilinearFunctionGlobal(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	// block and grid size
	int ngx = ((nx - 1) / LO_BW) + 1;
	int ngy = ((ny - 1) / LO_BH) + 1;

	dim3 dimGrid( ngx, ngy );
	dim3 dimBlock( LO_BW, LO_BH );

	//call warp method on gpu
	backwardRegistrationBilinearFunctionGlobalGpu<<<dimGrid, dimBlock>>>(in_g,
			flow1_g, flow2_g, out_g, constant_g, nx, ny, pitchf1_in,
			pitchf1_out, hx, hy);
}

void backwardRegistrationBilinearFunctionTex(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy)
{
	// ### Implement me, if you want ###
}

void forewardRegistrationBilinearAtomic(const float *flow1_g,
		const float *flow2_g, const float *in_g, float *out_g, int nx, int ny,
		int pitchf1)
{
	// ### Implement me ###
}

void gaussBlurSeparateMirrorGpu(float *in_g, float *out_g, int nx, int ny,
		int pitchf1, float sigmax, float sigmay, int radius, float *temp_g,
		float *mask)
{
	// ### Implement me ###
}

/*
 __global__ void resampleAreaParallelSeparate_x
 (
 const float * in_g,
 float * out_g,
 int nx,
 int ny,
 float hx,
 int pitchf1_in,
 float factor = 0.0f
 )
 {
 if( factor == 0.0f ) { factor = 1/hx; }

 int ix = threadIdx.x + blockIdx.x * blockDim.x;
 int iy = threadIdx.y + blockIdx.y * blockDim.y;

 int index = ix + iy * pitchf1_in; // global index for out image

 if( ix < nx && iy < ny)
 {
 // initialising out
 out_g[ index ] = 0.0f;

 float px = (float)ix * hx;

 float left = ceil(px) - px;
 if(left > hx) left = hx;

 float midx = hx - left;
 float right = midx - floorf(midx);
 midx = midx - right;

 if( left > 0.0f )
 {
 // using pitchf1_in instead of nx_orig in original code
 out_g[index] += in_g[ iy * pitchf1_in + (int) floor(px) ] * left * factor; // look out for conversion of coordinates
 px += 1.0f;
 }
 while(midx > 0.0f)
 {
 // using pitchf1_in instead of nx_orig in original code
 out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * factor;
 px += 1.0f;
 midx -= 1.0f;
 }
 if(right > RESAMPLE_EPSILON)
 {
 // using pitchf1_in instead of nx_orig in original code
 out_g[index] += in_g[ iy * pitchf1_in + (int)(floor(px))] * right * factor;
 }
 }
 }

 __global__ void resampleAreaParallelSeparate_y
 (
 const float * in_g,
 float * out_g,
 int nx,
 int ny,
 float hy,
 int pitchf1_out,
 float factor = 0.0f // need
 )
 {
 if(factor == 0.0f) factor = 1.0f/hy;

 int ix = threadIdx.x + blockIdx.x * blockDim.x;
 int iy = threadIdx.y + blockIdx.y * blockDim.y;

 int index = ix + iy * pitchf1_out; // global index for out image
 // used pitch instead  of blockDim.x

 if( ix < nx && iy < ny ) // guards
 {
 out_g[index] = 0.0f;

 float py = (float)iy * hy;
 float top = ceil(py) - py;

 if(top > hy) top = hy;
 float midy = hy - top;

 float bottom = midy - floorf(midy);
 midy = midy - bottom;

 if(top > 0.0f)
 {
 // using pitch for helper array since these all arrays have same pitch
 out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * top * factor;
 py += 1.0f;
 }
 while(midy > 0.0f)
 {
 out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * factor;
 py += 1.0f;
 midy -= 1.0f;
 }
 if(bottom > RESAMPLE_EPSILON)
 {
 out_g[index] += in_g[(int)(floor(py)) * pitchf1_out + ix ] * bottom * factor;
 }
 }
 }

 */

__global__ void resampleAreaParallelSeparateGpu_x(const float *in_g,
		float *out_g, int nx, int ny, float hx, int pitchf1_in, int pitchf1_out,
		float scalefactor = 0.0f)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// check if x is within the boundaries
	if (!(x < nx && y < ny))
	{
		return;
	}

	int p = y * pitchf1_out + x;
	// resampling in x
	if (scalefactor == 0.0f)
		scalefactor = 1.0f / hx;

	float px = (float) x * hx;
	float left = ceil(px) - px;
	if (left > hx)
		left = hx;
	float midx = hx - left;
	float right = midx - floorf(midx);
	midx = midx - right;

	out_g[p] = 0.0f;

	if (left > 0.0f)
	{
		out_g[p] += in_g[y * pitchf1_in + (int) (floor(px))] * left
				* scalefactor;
		px += 1.0f;
	}
	while (midx > 0.0f)
	{
		out_g[p] += in_g[y * pitchf1_in + (int) (floor(px))] * scalefactor;
		px += 1.0f;
		midx -= 1.0f;
	}
	if (right > RESAMPLE_EPSILON)
	{
		out_g[p] += in_g[y * pitchf1_in + (int) (floor(px))] * right
				* scalefactor;
	}

}

__global__ void resampleAreaParallelSeparateGpu_y(const float *in_g,
		float *out_g, int nx, int ny, float hy, int pitchf1, float scalefactor =
				0.0f)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// check if x is within the boundaries
	if (!(x < nx && y < ny))
	{
		return;
	}

	int p = y * pitchf1 + x;

	// resampling in y
	if (scalefactor == 0.0f)
		scalefactor = 1.0f / hy;

	float py = (float) y * hy;
	float top = ceil(py) - py;
	if (top > hy)
		top = hy;
	float midy = hy - top;
	float bottom = midy - floorf(midy);
	midy = midy - bottom;

	out_g[p] = 0.0f;

	if (top > 0.0f)
	{
		out_g[p] += in_g[(int) (floor(py)) * pitchf1 + x] * top * scalefactor;
		py += 1.0f;
	}
	while (midy > 0.0f)
	{
		out_g[p] += in_g[(int) (floor(py)) * pitchf1 + x] * scalefactor;
		py += 1.0f;
		midy -= 1.0f;
	}
	if (bottom > RESAMPLE_EPSILON)
	{
		out_g[p] += in_g[(int) (floor(py)) * pitchf1 + x] * bottom
				* scalefactor;
	}

}

void resampleAreaParallelSeparate(const float *in_g, float *out_g, int nx_in,
		int ny_in, int pitchf1_in, int nx_out, int ny_out, int pitchf1_out,
		float *help_g, float scalefactor = 1.0f)
{

	if (help_g == 0)
	{
		fprintf(stderr, "\nERROR: no helper array for resamling!");
		return;
	}

	// first use x_out and y_in
	int ngx = (nx_out % LO_BW) ? ((nx_out / LO_BW) + 1) : (nx_out / LO_BW);
	int ngy = (ny_in % LO_BH)  ? ((ny_in / LO_BH)+1) : (ny_in / LO_BH);
	dim3 dimGrid(ngx, ngy);
	dim3 dimBlock(LO_BW, LO_BH);
	float hx = (float) (nx_in) / (float) (nx_out);
	float hy = (float) (ny_in) / (float) (ny_out);

	resampleAreaParallelSeparateGpu_x<<<dimGrid, dimBlock>>>(in_g, help_g,
			nx_out, ny_in, hx, pitchf1_in, pitchf1_out,
			(float) (nx_out) / (float) (nx_in));

	catchkernel;

	// this cost us a lot of time -> resize grid to y_out
	ngy = (ny_out % LO_BH) ? ((ny_out / LO_BH)+1) : (ny_out / LO_BH);
	dimGrid = dim3(ngx, ngy);

	resampleAreaParallelSeparateGpu_y<<<dimGrid, dimBlock>>>(help_g, out_g,
			nx_out, ny_out, hy, pitchf1_out,
			scalefactor * (float) (ny_out) / (float) (ny_in));

	catchkernel;
}

/*
 void resampleAreaParallelSeparate(const float *in_g, float *out_g, int nx_in,
 int ny_in, int pitchf1_in, int nx_out, int ny_out, int pitchf1_out,
 float *help_g, float scalefactor)
 {
 // helper array is already allocated on the GPU as _b1, now help_g

 // can reduce no of blocks for first pass
 dim3 dimGrid((int)ceil((float)nx_out/LO_BW), (int)ceil((float)ny_out/LO_BH));
 dim3 dimBlock(LO_BW,LO_BH);

 float hx = (float) nx_in/ (float) nx_out;
 float factor = (float)(nx_out)/(float)(nx_in);
 resampleAreaParallelSeparate_x<<< dimGrid, dimBlock >>>( in_g, help_g, nx_out, ny_in, hx, pitchf1_in, factor);

 float hy = (float)(ny_in)/(float)(ny_out);
 factor = scalefactor*(float)(ny_out)/(float)(ny_in);
 resampleAreaParallelSeparate_y<<< dimGrid, dimBlock >>>( help_g, out_g, nx_out, ny_out, hy, pitchf1_out, factor );
 }
 */

void resampleAreaParallelSeparateAdjoined(const float *in_g, float *out_g,
		int nx_in, int ny_in, int pitchf1_in, int nx_out, int ny_out,
		int pitchf1_out, float *help_g, float scalefactor)
{
	// ### Implement me ###
}

__global__ void addKernel(const float *increment_g, float *accumulator_g,
		int nx, int ny, int pitchf1)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y * pitchf1 + x;
	
	if( x < nx && y < ny )
	{
		accumulator_g[idx] += increment_g[idx];
	}
}

__global__ void subKernel(const float *increment_g, float *accumulator_g,
		int nx, int ny, int pitchf1)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y * pitchf1 + x;
	
	if( x < nx && y < ny )
	{
		accumulator_g[idx] -= increment_g[idx];
	}
}

__global__ void setKernel(float *field_g, int nx, int ny, int pitchf1,
		float value)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y * pitchf1 + x;
	
	if( x < nx && y < ny )
	{
		field_g[idx] = value;
	}
}
