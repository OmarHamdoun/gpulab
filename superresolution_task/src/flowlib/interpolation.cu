#include "hip/hip_runtime.h"
#include <auxiliary/cuda_basic.cuh>
#include <linearoperations/linearoperations.cuh>
#include <auxiliary/debug.hpp>
#include <string>

#define IP_BW 16
#define IP_BH 16

#define IMAGES_TO_INTERPOLATE 20


// gpu warping kernel with global memory
__global__ void backwardRegistrationBilinearFunctionGlobalFactorGpu(const float *in_g,
		const float *flow1_g, const float *flow2_g, float *out_g,
		const float *constant_g, int nx, int ny, int pitchf1_in,
		int pitchf1_out, float hx, float hy, float factor)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// check if x is within the boundaries
	if (x < nx && y < ny)
	{
		const float xx = (float) x + factor * flow1_g[y * pitchf1_in + x] / hx;
		const float yy = (float) y + factor * flow2_g[y * pitchf1_in + x] / hy;

		int xxFloor = (int) floor(xx);
		int yyFloor = (int) floor(yy);

		int xxCeil = xxFloor == nx - 1 ? xxFloor : xxFloor + 1;
		int yyCeil = yyFloor == ny - 1 ? yyFloor : yyFloor + 1;

		float xxRest = xx - (float) xxFloor;
		float yyRest = yy - (float) yyFloor;

		//same weird expression as in cpp
		out_g[y * pitchf1_out + x] =
				(xx < 0.0f || yy < 0.0f || xx > (float) (nx - 1)
						|| yy > (float) (ny - 1)) ?
						constant_g[y * pitchf1_in + x] :
						(1.0f - xxRest) * (1.0f - yyRest)
								* in_g[yyFloor * pitchf1_in + xxFloor]
								+ xxRest * (1.0f - yyRest)
										* in_g[yyFloor * pitchf1_in + xxCeil]
								+ (1.0f - xxRest) * yyRest
										* in_g[yyCeil * pitchf1_in + xxFloor]
								+ xxRest * yyRest
										* in_g[yyCeil * pitchf1_in + xxCeil];

	}
}



void interpolateImages
	(
		float* image1_g,
		float* image2_g,
		float* u_g,
		float* v_g,
		int nx,
		int ny,
		int pitch
	)
{
	// block and grid size
	int gridsize_x = ((nx - 1) / IP_BW) + 1;
	int gridsize_y = ((ny - 1) / IP_BH) + 1;

	dim3 dimGrid( gridsize_x, gridsize_y );
	dim3 dimBlock( IP_BW, IP_BH );

	// allocate memory for file names
	char fileName[128];

	// allocate GPU memory for result images
	float* result_g;
	cuda_malloc2D( (void**)&result_g, nx, ny, 1, sizeof(float), &pitch );

	// reset result image
	setKernel <<< dimGrid, dimBlock >>> ( result_g, nx, ny, pitch, 0.0f );



	float stepSize = 1 / (float)(IMAGES_TO_INTERPOLATE + 1);
	float factor = stepSize;

	fprintf( stderr, "\nFlow factor: %f, step size: %f", factor, stepSize );

	for( int i = 0; i < IMAGES_TO_INTERPOLATE; ++i )
	{
		// warp image
		//foreward_warp_kernel_atomic_factor <<< dimGrid, dimBlock >>> (
		//		u_g, v_g, image1_g, result_g, nx, ny, pitch, factor );
		backwardRegistrationBilinearFunctionGlobalFactorGpu<<<dimGrid, dimBlock>>>(
				image2_g, u_g, v_g, result_g, image1_g, nx, ny, pitch,
					pitch, 1.0f, 1.0f, factor );

		// save interpolated image
		fprintf( stderr, "\nSaving interpolated image %d (factor %f)...", i, factor );
		snprintf( fileName, 128, "interpolation/interpolation_%02d.png", i );
		saveCudaImage( fileName, result_g, nx, ny, pitch, 1 );

		factor += stepSize;

	}



	// clean up
	cutilSafeCall( hipFree( result_g ) );
}
