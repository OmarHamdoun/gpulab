#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    superresolution.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * superresolution.cu
 *
 *  Created on: May 16, 2012
 *      Author: steinbrf
 */
#include "superresolution.cuh"
#include <stdio.h>
//#include <cutil.h>
//#include <cutil_inline.h>
#include <auxiliary/cuda_basic.cuh>
#include <vector>
#include <list>

//#include <linearoperations.cuh>
#include <linearoperations/linearoperations.cuh>

#include "superresolution_definitions.h"

#include <auxiliary/debug.hpp>


#ifdef DGT400
#define SR_BW 32
#define SR_BH 16
#else
#define SR_BW 16
#define SR_BH 16
#endif

#include <linearoperations/linearoperations.h>


extern __shared__ float smem[];

void computeSuperresolutionUngerGPU
(
		float *xi1_g, 							//! Dual Variable for TV regularization in X direction
		float *xi2_g,							//! Dual Variable for TV regularization in X direction
		float *temp1_g,							//! Helper array
		float *temp2_g,
		float *temp3_g,
		float *temp4_g,
		float *uor_g,							//! Field of overrelaxed primal variables
		float *u_g,								//! GPU memory for the result image
		std::vector<float*> &q_g,				//! Dual variables for L1 difference penalization
		std::vector<float*> &images_g,			//! Dual variables for L1 difference penalization
		std::list<FlowGPU> &flowsGPU,			//! GPU memory for the displacement fields
		int   &nx,								//! New High-Resolution Width
		int   &ny,								//! New High-Resolution Height
		int   &pitchf1,							//! GPU pitch (padded width) of the superresolution high-res fields
		int   &nx_orig,							//! Original Low-Resolution Width
		int   &ny_orig,							//! Original Low-Resolution Height
		int   &pitchf1_orig,					//! GPU pitch (padded width) of the original low-res images
		int   &oi,								//! Number of Iterations
		float &tau_p,							//! Primal Update Step Size
		float &tau_d,							//! Dual Update Step Size
		float &factor_tv,						//! The weight of Total Variation Penalization
		float &huber_epsilon,					//! Parameter for Huber norm regularization
		float &factor_rescale_x,				//! High-Resolution Width divided by Low-Resolution Width
		float &factor_rescale_y,				//! High-Resolution Height divided by Low-Resolution Height
		float &blur,							//! The amount of Gaussian Blur present in the degrading process
		float &overrelaxation,					//! Overrelaxation parameter in the range of [1,2]
		int   debug								//! Debug Flag, if activated the class produces Debug output.
)
{
	//### Implement me###
	fprintf(stderr,"\nComputing 1N Superresolution from %i Images on GPU",(int)_images_original.size());

	// replacing u by u_g ( pointer to resultant data)
	
	// TODO: KERNEL TO INITIALISE xi1_g & xi2_g to 0.0f
	// TODO: KERNEL TO INITIALISE u_g & uor_g to 64.0f
	
	// TODO: KERNEL TO SET q_g TO 0.0f, LOOPS _q.size() TIMES 
	
	float factorquad = factor_rescale_x*factor_rescale_y*factor_rescale_x*factor_rescale_y;
	float factor_degrade_update = pow(factorquad,CLIPPING_TRADEOFF_DEGRADE_1N);
	
	float factor_degrade_clipping = factorquad/factor_degrade_update;
	float huber_denom_degrade = 1.0f + huber_epsilon*tau_d/factor_degrade_clipping;

	float factor_tv_update = pow(_factor_tv,CLIPPING_TRADEOFF_TV);
	float factor_tv_clipping = factor_tv/factor_tv_update;
	float huber_denom_tv = 1.0f + huber_epsilon*tau_d/_factor_tv;
	
	for(int i=0;i<_iterations;i++)
	{
		fprintf(stderr," %i",i);

		//TODO: KERNEL FOR DUAL TV
		//dualTVHuber(_u_overrelaxed,_xi1,_xi2,_nx,_ny,factor_tv_update,factor_tv_clipping,huber_denom_tv,_tau_d);

		//DUAL DATA
		unsigned int k=0;
		
		// NEED TO INITIALISE A ITERATOR FOR ORIGINAL IMAGES
		//std::vector<cv::Mat*>::iterator image = _images_original.begin();
		
		// NEED TO SET A ITERATOR FOR FLOWS
		//std::list<Flow>::iterator flow = _flows.begin();
				
		// for( k=0; k< _(while image is not the last original image)__ ; k++ ) // TODO
		{
				// TODO: KERNEL BACKWARDREGISTRATIONBILINEARVALUE
				
		
				if(blur > 0.0f)
				{
					// TODO: KERNEL FOR GAUSSBLURSEPARATEMIRROR
				}
				else
				{
					// SWAP THE HELPER ARRAY POINTERS
					float *temp = temp1_g; temp1_g = temp2_g; temp2_g = temp;
				}
		
				if(factor_rescale_x > 1.0f || factor_rescale_y > 1.0f)
				{
					resampleAreaParallelSeparate(temp2_g, temp1_g, nx, ny,
												pitchf1, nx_orig, ny_orig,
												pitchf1_orig, temp4_g);
				}
				else
				{
					// SWAP THE HELPER ARRAY POINTERS
					float *temp = temp1_g; temp1_g = temp2_g; temp2_g = temp;
				}
				
				// TODO: KERNEL FOR dualL1Difference
				
				// USE k IN FOR LOOP
				//k++; 
		}
		
		// TODO: KERNEL TO SET 3RD HELPER ARRAY TO 0.00f
		
		k=0;
		image = _images_original.begin();
		flow = _flows.begin();
		// for( k=0; k< _(while image is not the last original image)__ ; k++ ) // TODO
		{
			if(factor_rescale_x > 1.0f || factor_rescale_y > 1.0f)
			{
				// TODO: WRITE KERNEL resampleAreaParallelizableSeparateAdjoined
			}
			else
			{
				// TODO: KERNEL TO COPY q_g[k] to temp1_g
			}
			if(blur > 0.0f)
			{
				// TODO: KERNEL FOR GAUSSBLURSEPARATEMIRROR
				// lookout for change in parameters, if any
			}
			else
			{
				// SWAP THE HELPER ARRAY POINTERS
				float *temp = temp1_g; temp1_g = temp2_g; temp2_g = temp;
			}
			
			// TODO: IMPLEMENT forewardRegistrationBilinear
			
			// TODO: KERNEL TO ADD 1ST TO 3RD HELPER ARRAY
		}	
		
		// TODO: IMPLMENT KERNEL primal1N
}





