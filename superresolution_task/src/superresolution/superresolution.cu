#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    superresolution.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * superresolution.cu
 *
 *  Created on: May 16, 2012
 *      Author: steinbrf
 */
#include "superresolution.cuh"
#include <stdio.h>
//#include <cutil.h>
//#include <cutil_inline.h>
#include <auxiliary/cuda_basic.cuh>
#include <vector>
#include <list>

//#include <linearoperations.cuh>
#include <linearoperations/linearoperations.cuh>

#include "superresolution_definitions.h"

#include <auxiliary/debug.hpp>


#ifdef DGT400
#define SR_BW 32
#define SR_BH 16
#else
#define SR_BW 16
#define SR_BH 16
#endif

//shared mem flags
#define SHARED_MEM 0

#include <linearoperations/linearoperations.h>

//TODO where the heck should this used
extern __shared__ float smem[];

//TODO write comment
// global memory version of dualL1Difference
__global__ void dualL1Difference_gm
(
    const float *primal,
    const float *constant,
    float *dual,
    int nx,
    int ny,
    int pitch,
    float factor_update,
    float factor_clipping,
    float huber_denom,
    float tau_d
    )
{
  const int x = threadIdx.x + blockDim.x * blockIdx.x;
  const int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < nx && y < ny)
  {
    int idx = x + pitch * y;
    dual[idx] = (dual[idx] + tau_d * factor_update * (primal[idx] - constant[idx]))
    		    / huber_denom;
    if (dual[idx] < -factor_clipping)
    {
    	dual[idx] = -factor_clipping;
    }

    if (dual[idx] > factor_clipping)
    {
    	dual[idx] = factor_clipping;
    }
  }
}

//TODO write comment
// shared memory version of primal1N
__global__ void dualL1Difference_sm
(
    const float *primal,
    const float *constant,
    float *dual,
    int nx,
    int ny,
    int pitch,
    float factor_update,
    float factor_clipping,
    float huber_denom,
    float tau_d
    )
{
  const int x = threadIdx.x + blockDim.x * blockIdx.x;
  const int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < nx && y < ny)
  {
	  //TODO implement
  }
}

//TODO write comment
// global memory version of primal1N
__global__ void primal1N_gm
(
    const float *xi1,
    const float *xi2,
    const float *degraded,
    float *u,
    float *uor,
    int nx,
    int ny,
    int pitch,
    float factor_tv_update,
    float factor_degrade_update,
    float tau_p,
    float overrelaxation
    )
{
  const int x = threadIdx.x + blockDim.x * blockIdx.x;
  const int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < nx && y < ny)
  {
    const int idx = y * pitch + x;
    float u_old = u[idx];
    float u_new = u[idx] + tau_p *
        (factor_tv_update * (xi1[idx] - (x == 0 ? 0.0f : xi1[idx - 1]) + xi2[idx] - (y == 0 ? 0.0f : xi2[idx - nx]))
            - factor_degrade_update * degraded[idx]);
    u[idx] = u_new;
    uor[idx] = overrelaxation * u_new + (1.0f - overrelaxation) * u_old;
  }
}

__global__ void primal1N_sm
(
    const float *xi1,
    const float *xi2,
    const float *degraded,
    float *u,
    float *uor,
    int nx,
    int ny,
    int pitch,
    float factor_tv_update,
    float factor_degrade_update,
    float tau_p,
    float overrelaxation
    )
{
  const int x = threadIdx.x + blockDim.x * blockIdx.x;
  const int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < nx && y < ny)
  {
   //TODO implement me
  }
}

//TODO write comment
// global memory version of primal1N
__global__ void dualTVHuber_gm
(
		float 	*uor_g,								// Field of overrelaxed primal variables
		float 	*xi1_g, 							// Dual Variable for TV regularization in X direction
		float 	*xi2_g,								// Dual Variable for TV regularization in Y direction
		int   	nx,									// New High-Resolution Width
		int   	ny,									// New High-Resolution Height
		int   	pitchf1,							// GPU pitch (padded width) of the superresolution high-res fields
		float   factor_update,
		float   factor_clipping,
		float   huber_denom,
		float   tau_d
)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	if( x < nx && y < ny ) // guards
	{
		//DONT FORGET TO USE THE PITCH
		
		int x1 = x + 1;
		if( x1 >= nx ){	x1 = nx-1; }	// at x boundary
		
		int y1 = y+1; 
		if( y1 >= ny ){ y1 = ny-1; }	// at y boundary
		
		// do xi1_g, xi2_g & uor_g have same pitch ? confirm - YES
	
		const int p = y * pitchf1 + x;
		
		float dx = (xi1_g[p] + tau_d * factor_update * (uor_g[y*pitchf1+x1] - uor_g[p])) /huber_denom;
		float dy = (xi2_g[p] + tau_d * factor_update * (uor_g[y1*pitchf1+x] - uor_g[p])) /huber_denom;
		float denom = sqrtf( dx * dx + dy * dy ) / factor_clipping;
		
		if(denom < 1.0f) denom = 1.0f;
		xi1_g[p] = dx / denom;
		xi2_g[p] = dy / denom;
	}
}

//TODO write comment
// shared memory version of primal1N
__global__ void dualTVHuber_sm
(
		float 	*uor_g,								// Field of overrelaxed primal variables
		float 	*xi1_g, 							// Dual Variable for TV regularization in X direction
		float 	*xi2_g,								// Dual Variable for TV regularization in Y direction
		int   	nx,									// New High-Resolution Width
		int   	ny,									// New High-Resolution Height
		int   	pitchf1,							// GPU pitch (padded width) of the superresolution high-res fields
		float   factor_update,
		float   factor_clipping,
		float   huber_denom,
		float   tau_d
)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < nx && y < ny ) // guards
	{
		//TODO implement me
	}
}

void computeSuperresolutionUngerGPU
(
		float *xi1_g, 							// Dual Variable for TV regularization in X direction
		float *xi2_g,							// Dual Variable for TV regularization in Y direction
		float *temp1_g,							// Helper array
		float *temp2_g,
		float *temp3_g,
		float *temp4_g,
		float *uor_g,							// Field of overrelaxed primal variables
		float *u_g,								// GPU memory for the result image
		std::vector<float*> &q_g,				// Dual variables for L1 difference penalization
		std::vector<float*> &images_g,			// Dual variables for L1 difference penalization
		std::list<FlowGPU> &flowsGPU,			// GPU memory for the displacement fields
												//   class FlowGPU { void clear(); float *u_g; float *v_g; int nx; int ny; }
		int   &nx,								// New High-Resolution Width
		int   &ny,								// New High-Resolution Height
		int   &pitchf1,							// GPU pitch (padded width) of the superresolution high-res fields
		int   &nx_orig,							// Original Low-Resolution Width
		int   &ny_orig,							// Original Low-Resolution Height
		int   &pitchf1_orig,					// GPU pitch (padded width) of the original low-res images
		int   &oi,								// Number of Iterations
		float &tau_p,							// Primal Update Step Size
		float &tau_d,							// Dual Update Step Size
		float &factor_tv,						// The weight of Total Variation Penalization
		float &huber_epsilon,					// Parameter for Huber norm regularization
		float &factor_rescale_x,				// High-Resolution Width divided by Low-Resolution Width
		float &factor_rescale_y,				// High-Resolution Height divided by Low-Resolution Height
		float &blur,							// The amount of Gaussian Blur present in the degrading process
		float &overrelaxation,					// Overrelaxation parameter in the range of [1,2]
		int   debug								// Debug Flag, if activated the class produces Debug output.
)
{
	// replacing u by u_g ( pointer to resultant data)
	
	// grid and block dimensions
	int ngx = ((nx - 1) / SR_BW) + 1;
	int ngy = ((ny - 1) / SR_BH) + 1;
	dim3 dimGrid ( ngx, ngy );
	dim3 dimBlock ( SR_BW, SR_BH );
	
	// initialise xi1_g and xi2_g to zero
	setKernel <<<dimGrid, dimBlock>>>( xi1_g, nx, ny, pitchf1, 0.0f );
	setKernel <<<dimGrid, dimBlock>>>( xi2_g, nx, ny, pitchf1, 0.0f );
	
	// initialise u_g and uor_g to 64.0f
	setKernel <<<dimGrid, dimBlock>>>( u_g,   nx, ny, pitchf1, 64.0f );
	setKernel <<<dimGrid, dimBlock>>>( uor_g, nx, ny, pitchf1, 64.0f );
	
	// initialise all elements of q_g to zero
	for(unsigned int k = 0; k < q_g.size(); k++ )
	{
		setKernel <<<dimGrid, dimBlock>>>( q_g[k], nx, ny, pitchf1, 0.0f );
	}
	
	float factorquad              = factor_rescale_x * factor_rescale_x * factor_rescale_y * factor_rescale_y;
	float factor_degrade_update   = pow( factorquad, CLIPPING_TRADEOFF_DEGRADE_1N );
	
	float factor_degrade_clipping = factorquad / factor_degrade_update;
	float huber_denom_degrade     = 1.0f + huber_epsilon * tau_d / factor_degrade_clipping;

	float factor_tv_update        = pow( factor_tv, CLIPPING_TRADEOFF_TV );
	float factor_tv_clipping      = factor_tv / factor_tv_update;
	float huber_denom_tv          = 1.0f + huber_epsilon * tau_d / factor_tv;
	
	for(int i=0;i<oi;i++)
	{
		fprintf(stderr," %i",i);

		// TODO: KERNEL FOR DUAL TV
		int xBlocks = ( nx % SR_BW ) ? ( nx / SR_BW) + 1 : ( nx / SR_BW );
		int yBlocks = ( ny % SR_BH ) ? ( ny / SR_BH) + 1 : ( ny / SR_BH );
		
		dim3 dimGrid( xBlocks, yBlocks );
		dim3 dimBlock( SR_BW, SR_BH );
		

#ifdef SHARED_MEM
		dualTVHuber_sm<<<dimGrid,dimBlock>>>
				(uor_g,xi1_g,xi2_g,nx,ny,pitchf1,factor_tv_update,factor_tv_clipping,huber_denom_tv,tau_d);
#else
		dualTVHuber_gm<<<dimGrid,dimBlock>>>
				(uor_g,xi1_g,xi2_g,nx,ny,pitchf1,factor_tv_update,factor_tv_clipping,huber_denom_tv,tau_d);
#endif


		// DUAL DATA		
		
		// iterating over all images
		std::vector<float*>::iterator image = images_g.begin();
		std::list<FlowGPU>::iterator flow   = flowsGPU.begin();
		for( unsigned int k = 0; image != images_g.end() && flow != flowsGPU.end() && k < q_g.size(); ++k, ++flow, ++image )		
		{
				// TODO: KERNEL BACKWARDREGISTRATIONBILINEARVALUE
				
				if( blur > 0.0f )
				{
					// TODO: KERNEL FOR GAUSSBLURSEPARATEMIRROR
				}
				else
				{
					// swap the helper array pointers
					float *temp = temp1_g;
					temp1_g = temp2_g;
					temp2_g = temp;
				}
		
				if( factor_rescale_x > 1.0f || factor_rescale_y > 1.0f )
				{
					resampleAreaParallelSeparate(temp2_g, temp1_g, nx, ny,
												pitchf1, nx_orig, ny_orig,
												pitchf1_orig, temp4_g);
				}
				else
				{
					// swap the helper array pointers
					float *temp = temp1_g;
					temp1_g = temp2_g;
					temp2_g = temp;
				}
				
#ifdef SHARED_MEM
		dualL1Difference_sm<<<dimGrid,dimBlock>>>
				(uor_g,xi1_g,xi2_g,nx,ny,pitchf1,factor_tv_update,factor_tv_clipping,huber_denom_tv,tau_d);
#else
		dualL1Difference_gm<<<dimGrid, dimBlock>>>(temp1_g, *image, q_g[k], nx_orig, ny_orig, pitchf1_orig,
					          factor_degrade_update, factor_degrade_clipping, huber_denom_degrade, tau_d);
#endif


		}
		
		// set 3rd helper array to zero
		setKernel <<<dimGrid, dimBlock>>>( temp3_g, nx, ny, pitchf1, 0.0f );

		// iterating over all images
		image = images_g.begin();
		flow   = flowsGPU.begin();
		for( unsigned int k = 0; image != images_g.end() && flow != flowsGPU.end() && k < q_g.size(); ++k, ++flow, ++image )
		{
			if( factor_rescale_x > 1.0f || factor_rescale_y > 1.0f )
			{
				// TODO: WRITE KERNEL resampleAreaParallelizableSeparateAdjoined
				//CPU// resampleAreaParallelizableSeparateAdjoined(_q[k],_help1,_nx_orig,_ny_orig,_nx,_ny,_help4);
				
				/*
				 *  Assuming it resamples a image from q_g[k] from size nx_orig*ny_orig(pitch- pitchf1_orig) to
				 *  new size nx*ny (pitch- pitchf1) and stores it in temp1_g with the help of helper array
				 *  temp4_g. The image in temp1_g is then used for gaussian Blurring
				 */
				resampleAreaParallelSeparateAdjoined( q_g[k], temp1_g, nx_orig, ny_orig, pitchf1_orig, nx, ny, pitchf1, temp4_g );
			}
			else
			{
				// copy q_g[k] to temp1_g
				hipMemcpy( temp1_g, q_g[k], ny * pitchf1, hipMemcpyDeviceToDevice ); 	
			}
			
			if( blur > 0.0f )
			{
				// TODO: KERNEL FOR GAUSSBLURSEPARATEMIRROR (lookout for change in parameters, if any)
			}
			else
			{
				// swap the helper array pointers
				float *temp = temp1_g;
				temp1_g = temp2_g;
				temp2_g = temp;
			}
			
			// foreward warping
			forewardRegistrationBilinearAtomic (
					flow->u_g, flow->v_g,
					temp2_g, temp1_g,
					nx, ny,
					pitchf1
				);
			
			// add 1st to 3rd helper array
			addKernel <<<dimGrid, dimBlock>>>( temp1_g, temp3_g, nx, ny, pitchf1 );
		}
		
#ifdef SHARED_MEM
		primal1N_sm<<< dimGrid, dimBlock>>>(xi1_g, xi2_g, temp3_g, u_g, uor_g, nx, ny, pitchf1, factor_tv_update, factor_degrade_update, tau_p, overrelaxation);
#else
	    primal1N_gm<<< dimGrid, dimBlock>>>(xi1_g, xi2_g, temp3_g, u_g, uor_g, nx, ny, pitchf1, factor_tv_update, factor_degrade_update, tau_p, overrelaxation);
#endif

	}	
}
